#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
#include "geneCuda.h"

#define CONSERVATIVE_GROUPS "NDEQ|NEQK|STA|MILV|QHRK|NHQK|FYW|HY|MILF"
#define SEMI_CONSERVATIVE_GROUPS "SAG|ATV|CSA|SGND|STPA|STNK|NEQHRK|NDEQHK|SNDEQK|HFY|FVLIM"

// a macro for quick aloocating and copying ro device memory 
#define cudeMemloc(dst, src, size, type) {\
				size_t arrSize = size * sizeof(type);\
				hipMalloc((void**)&dst, arrSize);\
				hipMemcpy(dst, src, arrSize, hipMemcpyHostToDevice); }\
				

__device__ __host__ int checkConsGroups(char letter1, char letter2, const char* conservativeGroups)
{
	int counter = 0;
	for (int i = 0; conservativeGroups[i]; i++)
	{
		if (conservativeGroups[i] == '|')
		{
			counter = 0;
			continue;
		}
		if (letter1 == conservativeGroups[i] || letter2 == conservativeGroups[i])
			counter++;
		if (counter >= 2)
			return 1;
	}
	return 0;
}

__device__ __host__ int checkSemiConsGroups(char letter1, char letter2, const char* semiConservativeGroups)
{
	int counter = 0;
	for (int i = 0; semiConservativeGroups[i]; i++)
	{
		if (semiConservativeGroups[i] == '|')
		{
			counter = 0;
			continue;
		}
		if (letter1 == semiConservativeGroups[i] || letter2 == semiConservativeGroups[i])
			counter++;
		if (counter >= 2)
			return 1;
	}
	return 0;
}

// a method to compare a pair of letters, returns '*' if in same conservative group, ':' if in same semi conservative group, otherwise ' '
__device__ __host__ char comparePairInternal(char letter1, char letter2)
{
	// check if letters are equal
	if (letter1 == letter2)
		return '*';
	// else check groups
	if (checkConsGroups(letter1, letter2, CONSERVATIVE_GROUPS))
		return ':';
	if (checkSemiConsGroups(letter1, letter2, SEMI_CONSERVATIVE_GROUPS))
		return '.';
	// return ' ' if no common group was found
	return ' ';
}

char comparePair(char letter1, char letter2)
{
	return comparePairInternal(letter1, letter2);
}

__global__ void findOptimalMutationInternal(char** map, char* seq1, char* mutant, float* weights, float* results, int* params)
{
	// get the unique id(e.g assigned letter)
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	int bestIndex = -1;

	int offset = params[0];
	int direction = params[1];

	// iterate over every possiable mutation of the assigned letter and pick the optimal one
	for (int j = 0; map[i][j]; j++)
	{
		float newValue = direction;
		char sign = comparePairInternal(map[i][j], seq1[i + offset]);
		if (sign == '*')
			newValue *= weights[0];
		else if (sign == ':')
			newValue *= -weights[1];
		else if (sign == '.')
			newValue *= -weights[2];
		else
			newValue *= -weights[3];
		
		if (bestIndex == -1 || results[i] < newValue)
		{
			bestIndex = j;
			results[i] = newValue;
		}
	}
	
	mutant[i] = map[i][bestIndex];
	
}

float findOptimalMutation(char** map, char* seq1, int len1, char* mutant, int len2, float* weights, int offset, int direction)
{
	// copy the input data into device memory
	char* tempMap[len2];
	for (int i = 0; i < len2; i++)
		cudeMemloc(tempMap[i], map[i], len2, char);
	
	char** cudaMap;
	cudeMemloc(cudaMap, tempMap, len2, char*);
	
	char* cudaSeq1;
	cudeMemloc(cudaSeq1, seq1, len1, char);
	
	char* cudaMutant;
	hipMalloc((void**)&cudaMutant, len2);
	
	float* cudaWeights;
	cudeMemloc(cudaWeights, weights, 4, float);
	
	float* cudaResults;
	size_t size = len2 * sizeof(float);
	hipMalloc((void**)&cudaResults, size);
	
	int params[2] = {offset, direction};
	
	int* cudaParmas;
	cudeMemloc(cudaParmas, params, 2, int);
	
	// start the kernal
	int blocksPerGrid = 1;
	int threadsPerBlock = len2;
	findOptimalMutationInternal<<<blocksPerGrid, threadsPerBlock>>>(cudaMap, cudaSeq1, cudaMutant, cudaWeights, cudaResults, cudaParmas);

	// get the results from the kernal
	hipMemcpy(mutant, cudaMutant, len2, hipMemcpyDeviceToHost);
	
	mutant[len2] = '\0';
	
	float results[len2];
	hipMemcpy(results, cudaResults, size, hipMemcpyDeviceToHost);
	
	// compute sum of results and return value
	float output = 0;
	for (int i = 0; i < len2; i++)
		output += results[i];
		
	return output;
}




